#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <THC/THCAtomics.cuh>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

#include "forward_warp.h"
using at::native::detail::GridSamplerInterpolation;

static __forceinline__ __device__ 
int get_im_index(
    const int b,
    const int c,
    const int h,
    const int w,
    const size_t C,
    const size_t H,
    const size_t W) {
  return b*C*H*W + c*H*W + h*W + w;
}

template <typename scalar_t>
__global__ void forward_warp_cuda_forward_kernel(
    const int total_step,
    const scalar_t* im0,
    const scalar_t* flow,
    scalar_t* im1,
    scalar_t* sort,
    const int B,
    const int C,
    const int H,
    const int W,
    const GridSamplerInterpolation interpolation_mode) {
  CUDA_KERNEL_LOOP(index, total_step) {
    const int b = index / (H * W);
    const int h = (index-b*H*W) / W;
    const int w = index % W;
    const scalar_t x = (scalar_t)w + flow[index*2+0];
    const scalar_t y = (scalar_t)h + flow[index*2+1];
    if (interpolation_mode == GridSamplerInterpolation::Bilinear) {
      const int x_f = static_cast<int>(::floor(x));
      const int y_f = static_cast<int>(::floor(y));
      const int x_c = x_f + 1;
      const int y_c = y_f + 1;
      if(x_f>=0 && x_c<W && y_f>=0 && y_c<H){
        const scalar_t nw_k = (x_c - x) * (y_c - y);
        const scalar_t ne_k = (x - x_f) * (y_c - y);
        const scalar_t sw_k = (x_c - x) * (y - y_f);
        const scalar_t se_k = (x - x_f) * (y - y_f);
        const scalar_t* im0_p = im0+get_im_index(b, 0, h, w, C, H, W);
        scalar_t* im1_p = im1+get_im_index(b, 0, y_f, x_f, C, H, W);
        scalar_t* sort_p = sort+get_im_index(b, 0, y_f, x_f, C, H, W);
        for (int c = 0; c < C; ++c, im0_p+=H*W, im1_p+=H*W, sort_p+=H*W){
          const scalar_t curr_sort = *sort_p;
          if (curr_sort < nw_k) {
            *sort_p = nw_k;
            *im1_p = nw_k * (*im0_p);
          }
          if (curr_sort < ne_k) {
            *sort_p = ne_k;
            *(im1_p+1) = ne_k * (*im0_p);
          }
          if (curr_sort < sw_k) {
            *sort_p = sw_k;
            *(im1_p+W) = sw_k * (*im0_p);
          }
          if (curr_sort < se_k) {
            *sort_p = se_k;
            *(im1_p+W+1) = se_k * (*im0_p);
          }
        }
      }
    } 
    else if (interpolation_mode == GridSamplerInterpolation::Nearest) {
      const int x_nearest = static_cast<int>(::round(x));
      const int y_nearest = static_cast<int>(::round(y));
      if(x_nearest>=0 && x_nearest<W && y_nearest>=0 && y_nearest<H){
        const scalar_t* im0_p = im0+get_im_index(b, 0, h, w, C, H, W);
        scalar_t* im1_p = im1+get_im_index(b, 0, y_nearest, x_nearest, C, H, W);
        scalar_t* sort_p = sort+get_im_index(b, 0, y_nearest, x_nearest, C, H, W);
        for (int c = 0; c < C; ++c, im0_p += H*W, im1_p += H*W, sort_p += H*W) {
          const scalar_t curr_sort = *sort_p;
          if (curr_sort < 1) {
            *sort_p = 1;
            *im1_p = *im0_p;
          }
        }
      }
    }
  }
}

template <typename scalar_t>
__global__ void forward_warp_cuda_backward_kernel(
    const int total_step,
    const scalar_t* grad_output,
    const scalar_t* im0,
    const scalar_t* flow,
    scalar_t* im0_grad,
    scalar_t* flow_grad,
    const int B,
    const int C,
    const int H,
    const int W,
    const GridSamplerInterpolation interpolation_mode) {
  CUDA_KERNEL_LOOP(index, total_step) {
    const int b = index / (H * W);
    const int h = (index-b*H*W) / W;
    const int w = index % W;
    const scalar_t x = (scalar_t)w + flow[index*2+0];
    const scalar_t y = (scalar_t)h + flow[index*2+1];
    if (interpolation_mode == GridSamplerInterpolation::Bilinear) {
      const int x_f = static_cast<int>(::floor(x));
      const int y_f = static_cast<int>(::floor(y));
      const int x_c = x_f + 1;
      const int y_c = y_f + 1;
      if(x_f>=0 && x_c<W && y_f>=0 && y_c<H){
        const scalar_t nw_k = (x_c - x) * (y_c - y);
        const scalar_t sw_k = (x_c - x) * (y - y_f);
        const scalar_t ne_k = (x - x_f) * (y_c - y);
        const scalar_t se_k = (x - x_f) * (y - y_f);
        scalar_t flow_grad_x = 0;
        scalar_t flow_grad_y = 0;
        scalar_t* im0_grad_p = im0_grad+get_im_index(b, 0, h, w, C, H, W);
        for (int c = 0; c < C; ++c, im0_grad_p+=H*W){
          const scalar_t nw_grad = grad_output[get_im_index(b, c, y_f, x_f, C, H, W)];
          const scalar_t ne_grad = grad_output[get_im_index(b, c, y_f, x_c, C, H, W)];
          const scalar_t sw_grad = grad_output[get_im_index(b, c, y_c,
